/*
 * Copyright 2011-2015 NVIDIA Corporation. All rights reserved
 *
 * Sample CUPTI app to print a trace of CUDA API and GPU activity
 */ 
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <string.h>
#include <vector>
#include <thread>

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define COMPUTE_N 50000
#define THREADS 8
#define ITERATIONS 1000

#ifdef TRACER
extern void initTrace(void);
extern void finiTrace(void);
#endif

// Kernels
__global__ void 
VecAdd(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

__global__ void 
VecSub(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] - B[i];
}

static void
do_pass(hipStream_t stream)
{
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;
  size_t size = COMPUTE_N * sizeof(int);
  int threadsPerBlock = 256;
  int blocksPerGrid = 0;
  
  // Allocate input vectors h_A and h_B in host memory
  // don't bother to initialize
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);
  
  // Allocate vectors in device memory
  RUNTIME_API_CALL(hipMalloc((void**)&d_A, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_B, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_C, size));

  RUNTIME_API_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream));
  RUNTIME_API_CALL(hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream));

  blocksPerGrid = (COMPUTE_N + threadsPerBlock - 1) / threadsPerBlock;
  VecAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, COMPUTE_N);
  VecSub<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, COMPUTE_N);
    
  RUNTIME_API_CALL(hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream));

/*
  if (stream == 0)
    RUNTIME_API_CALL(cudaDeviceSynchronize());
  else
    RUNTIME_API_CALL(cudaStreamSynchronize(stream));
*/

  free(h_A);
  free(h_B);
  free(h_C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

void myfunc(int deviceNum, hipStream_t streamCommon) {
  hipDevice_t device;  
  char deviceName[32];
  DRIVER_API_CALL(hipDeviceGet(&device, deviceNum));
  DRIVER_API_CALL(hipDeviceGetName(deviceName, 32, device));
  printf("Device Name: %s\n", deviceName);

  RUNTIME_API_CALL(hipSetDevice(deviceNum));
  hipStream_t stream0;
  RUNTIME_API_CALL(hipStreamCreate(&stream0));

  for (int i = 0 ; i < ITERATIONS ; i++) {
      // do pass default stream
      do_pass(0);
      // do pass with user stream
      do_pass(stream0);
      // do pass with common stream
      hipStream_t stream2;
      // do pass with temporary stream
      RUNTIME_API_CALL(hipStreamCreate(&stream2));
      do_pass(stream2);
      RUNTIME_API_CALL(hipStreamDestroy(stream2));
  }
  //cudaDeviceSynchronize();
  RUNTIME_API_CALL(hipStreamDestroy(stream0));
}

int
main(int argc, char *argv[])
{
  int deviceNum = 0, devCount = 0;

  printf("cuInit()...\n");
  DRIVER_API_CALL(hipInit(0));
  
  printf("cuGetDeviceCount()...\n");
  RUNTIME_API_CALL(hipGetDeviceCount(&devCount));

  hipStream_t stream0;
  printf("cuSetDevice()...\n");
  RUNTIME_API_CALL(hipSetDevice(deviceNum));
  printf("cuStreamCreate()...\n");
  RUNTIME_API_CALL(hipStreamCreate(&stream0));

  printf("spawning threads...\n");
  std::vector<std::thread> threads;
  for (int index=0; index<THREADS; index++) {
    std::thread t(myfunc,deviceNum,stream0);
    threads.push_back(std::move(t));
  }

  printf("joining threads...\n");
  for (int index=0; index<THREADS; index++) {
    threads[index].join();
  }

  printf("Destrying main stream...\n");
  RUNTIME_API_CALL(hipDeviceSynchronize());
  RUNTIME_API_CALL(hipSetDevice(deviceNum));
  RUNTIME_API_CALL(hipStreamDestroy(stream0));

  return 0;
}

