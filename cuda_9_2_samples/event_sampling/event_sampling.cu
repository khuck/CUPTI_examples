#include "hip/hip_runtime.h"
/*
 * Copyright 2011-2017 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain profiler
 * event values by sampling.
 */


#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <cupti_events.h>

#ifdef _WIN32
#include <windows.h>
#else
#include <unistd.h>
#include <pthread.h>
#endif

#define CHECK_CU_ERROR(err, cufunc)                                     \
  if (err != hipSuccess)                                              \
    {                                                                   \
      printf ("Error %d for CUDA Driver API function '%s'.\n",          \
              err, cufunc);                                             \
      exit(-1);                                                         \
    }

#define CHECK_CUPTI_ERROR(err, cuptifunc)                       \
  if (err != CUPTI_SUCCESS)                                     \
    {                                                           \
      const char *errstr;                                       \
      cuptiGetResultString(err, &errstr);                       \
      printf ("%s:%d:Error %s for CUPTI API function '%s'.\n",  \
              __FILE__, __LINE__, errstr, cuptifunc);           \
      exit(-1);                                                 \
    }

#define EVENT_NAME "inst_executed"
#define N 100000
#define ITERATIONS 2000
#define SAMPLE_PERIOD_MS 50

// used to signal from the compute thread to the sampling thread
static volatile int testComplete = 0;

static hipCtx_t context;
static hipDevice_t device;
static const char *eventName;

// Device code
__global__ void VecAdd(const int* A, const int* B, int* C, int size)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  for(int n = 0 ; n < 100; n++) {
    if (i < size)
      C[i] = A[i] + B[i];
  }
}

static void
initVec(int *vec, int n)
{
  for (int i=0; i< n; i++)
    vec[i] = i;
}

void *
sampling_func(void *arg)
{
  CUptiResult cuptiErr;
  CUpti_EventGroup eventGroup;
  CUpti_EventID eventId;
  size_t bytesRead, valueSize;
  uint32_t numInstances = 0, j = 0;
  uint64_t *eventValues = NULL, eventVal = 0;
  uint32_t profile_all = 1;

  cuptiErr = cuptiSetEventCollectionMode(context,
                                         CUPTI_EVENT_COLLECTION_MODE_CONTINUOUS);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiSetEventCollectionMode");

  cuptiErr = cuptiEventGroupCreate(context, &eventGroup, 0);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupCreate");

  cuptiErr = cuptiEventGetIdFromName(device, eventName, &eventId);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGetIdFromName");

  cuptiErr = cuptiEventGroupAddEvent(eventGroup, eventId);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupAddEvent");

  cuptiErr = cuptiEventGroupSetAttribute(eventGroup,
                                         CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
                                         sizeof(profile_all), &profile_all);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupSetAttribute");

  cuptiErr = cuptiEventGroupEnable(eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupEnable");

  valueSize = sizeof(numInstances);
  cuptiErr = cuptiEventGroupGetAttribute(eventGroup,
                                         CUPTI_EVENT_GROUP_ATTR_INSTANCE_COUNT,
                                         &valueSize, &numInstances);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupGetAttribute");

  bytesRead = sizeof(uint64_t) * numInstances;
  eventValues = (uint64_t *) malloc(bytesRead);
  if (eventValues == NULL) {
      printf("%s:%d: Failed to allocate memory.\n", __FILE__, __LINE__);
      exit(-1);
  }

  while (!testComplete) {
    cuptiErr = cuptiEventGroupReadEvent(eventGroup,
                                        CUPTI_EVENT_READ_FLAG_NONE,
                                        eventId, &bytesRead, eventValues);
    CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupReadEvent");
    if (bytesRead != (sizeof(uint64_t) * numInstances)) {
      printf("Failed to read value for \"%s\"\n", eventName);
      exit(-1);
    }

    for (j = 0; j < numInstances; j++) {
      eventVal += eventValues[j];
    }
    printf("%s: %llu\n", eventName, (unsigned long long)eventVal);
#ifdef _WIN32
    Sleep(SAMPLE_PERIOD_MS);
#else
    usleep(SAMPLE_PERIOD_MS * 1000);
#endif
  }

  cuptiErr = cuptiEventGroupDisable(eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDisable");

  cuptiErr = cuptiEventGroupDestroy(eventGroup);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDestroy");

  free(eventValues);
  return NULL;
}

static void
compute(int iters)
{
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int sum, i;
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;

  // Allocate input vectors h_A and h_B in host memory
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel (multiple times to make sure we have time for
  // sampling)
  threadsPerBlock = 256;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  for (i = 0; i < iters; i++) {
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
  }


  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // Verify result
  for (i = 0; i < N; ++i) {
    sum = h_A[i] + h_B[i];
    if (h_C[i] != sum) {
      printf("kernel execution FAILED\n");
      exit(-1);
    }
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);
}

int
main(int argc, char *argv[])
{
#ifdef _WIN32
  HANDLE hThread;
#else
  int status;
  pthread_t pThread;
#endif
  hipError_t err;
  CUptiResult cuptiErr;
  int deviceNum;
  int deviceCount;
  char deviceName[32];
  CUpti_DeviceAttributeDeviceClass deviceClass;
  size_t deviceClassSize = sizeof deviceClass;

  printf("Usage: %s [device_num] [event_name]\n", argv[0]);

  err = hipInit(0);
  CHECK_CU_ERROR(err, "hipInit");

  err = hipGetDeviceCount(&deviceCount);
  CHECK_CU_ERROR(err, "hipGetDeviceCount");

  if (deviceCount == 0) {
    printf("There is no device supporting CUDA.\n");
    exit(-1);
  }

  if (argc > 1)
    deviceNum = atoi(argv[1]);
  else
    deviceNum = 0;
  printf("CUDA Device Number: %d\n", deviceNum);

  err = hipDeviceGet(&device, deviceNum);
  CHECK_CU_ERROR(err, "hipDeviceGet");

  err = hipDeviceGetName(deviceName, 32, device);
  CHECK_CU_ERROR(err, "hipDeviceGetName");

  printf("CUDA Device Name: %s\n", deviceName);

  cuptiErr = cuptiDeviceGetAttribute(device, CUPTI_DEVICE_ATTR_DEVICE_CLASS, &deviceClassSize, &deviceClass);
  CHECK_CUPTI_ERROR(cuptiErr, "cuptiDeviceGetAttribute");

  if (deviceClass != CUPTI_DEVICE_ATTR_DEVICE_CLASS_TESLA) {
    printf("Sample uses event collection mode _CONTINUOUS which is supported only on Tesla GPUs.\n");
    return 0;
  }

  if (argc > 2) {
    eventName = argv[2];
  }
  else {
    eventName = EVENT_NAME;
  }

  err = hipCtxCreate(&context, 0, device);
  CHECK_CU_ERROR(err, "hipCtxCreate");


  testComplete = 0;

  printf("Creating sampling thread\n");
#ifdef _WIN32
  hThread = CreateThread(NULL, 0, (LPTHREAD_START_ROUTINE) sampling_func,
                         NULL, 0, NULL );
  if (!hThread) {
    printf("CreateThread failed\n");
    exit(-1);
  }
#else
  status = pthread_create(&pThread, NULL, sampling_func, NULL);
  if (status != 0) {
    perror("pthread_create");
    exit(-1);
  }
#endif

  // run kernel while sampling
  compute(ITERATIONS);

  // "signal" the sampling thread to exit and wait for it
  testComplete = 1;
#ifdef _WIN32
  WaitForSingleObject(hThread, INFINITE);
#else
  pthread_join(pThread, NULL);
#endif

  hipDeviceSynchronize();
  return 0;
}

